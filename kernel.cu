#include "hip/hip_runtime.h"
/*
* ARQUITECTURA DE COMPUTADORES
* Hecho por: Adrián Zamora Sánchez y Adrián Alcalde Alzaga
* Ejercicio: Entregable 2 de CUDA
* Descripción: Dibujar un tablero de ajedrez utilizando un kernel bidimensional de bloques de 16x16 hilos 
* y donde cada hilo se encargue de generar un pixel de la imagen final
*/

// includes
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <iostream>
#include <math.h>
#include <time.h>
#include <>
#include "gpu_bitmap.h"
// defines
#define NUMHILOS 16 //hilos del programa
#define ANCHO 512 // Dimension horizontal
#define ALTO 512 // Dimension vertical

void propiedades_Device(int deviceID)
{
	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, deviceID);

	// calculo del numero de cores (SP)
	int cudaCores = 0;
	int SM = deviceProp.multiProcessorCount;
	int maxThreads = deviceProp.maxThreadsPerBlock;
	int major = deviceProp.major;
	int minor = deviceProp.minor;
	const char* archName;

	switch (major)
	{
	case 1:
		//TESLA
		archName = "TESLA";
		cudaCores = 8;
		break;
	case 2:
		//FERMI
		archName = "FERMI";
		if (minor == 0)
			cudaCores = 32;
		else
			cudaCores = 48;
		break;
	case 3:
		//KEPLER
		archName = "KEPLER";
		cudaCores = 192;
		break;
	case 5:
		//MAXWELL
		archName = "MAXWELL";
		cudaCores = 128;
		break;
	case 6:
		//PASCAL
		archName = "PASCAL";
		cudaCores = 64;
		break;
	case 7:
		//VOLTA(7.0) //TURING(7.5)
		cudaCores = 64;
		if (minor == 0)
			archName = "VOLTA";
		else
			archName = "TURING";
		break;
	case 8:
		// AMPERE
		archName = "AMPERE";
		cudaCores = 64;
		break;
	case 9:
		//HOPPER
		archName = "HOPPER";
		cudaCores = 64;
		break;
	default:
		//ARQUITECTURA DESCONOCIDA
		archName = "DESCONOCIDA";
	}

	int rtV;
	hipRuntimeGetVersion(&rtV);

	// presentacion de propiedades
	printf("***************************************************\n");
	printf("DEVICE %d: %s\n", deviceID, deviceProp.name);
	printf("***************************************************\n");
	printf("> CUDA Toolkit\t\t\t: %d.%d\n", rtV / 1000, (rtV % 1000) / 10);
	printf("> Arquitectura CUDA\t\t: %s\n", archName);
	printf("> Capacidad de Computo\t\t: %d.%d\n", major, minor);
	printf("> No. MultiProcesadores\t\t: %d\n", SM);
	printf("> No. Nucleos CUDA (%dx%d)\t: %d\n", cudaCores, SM, cudaCores * SM);
	printf("> Memoria Global (total)\t: %u MiB\n", deviceProp.totalGlobalMem / (1024 * 1024));
}


// GLOBAL: funcion llamada desde el host y ejecutada en el device (kernel)
__global__ void kernel(unsigned char* imagen)
{
	// ** Kernel bidimensional multibloque **
	//
	// coordenada horizontal de cada hilo
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	// coordenada vertical de cada hilo
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	// indice global de cada hilo (indice lineal para acceder a la memoria)
	int myID = x + y * blockDim.x * gridDim.x;
	// cada hilo obtiene la posicion de su pixel
	int miPixel = myID * 4;

	int tableroX = x / 64; // Cada cuadrado en el tablero tiene 64 píxeles en la dimensión x
	int tableroY = y / 64; // Cada cuadrado en el tablero tiene 64 píxeles en la dimensión y
	int tablero = (tableroX % 2) + (tableroY % 2); //posicion de cada pixel dependiendo de si es par o impar
	// cada hilo rellena los 4 canales de su pixel con un valor arbitrario
	if (tablero % 2 == 0)
	{
		imagen[miPixel + 0] = 0; // canal R
		imagen[miPixel + 1] = 0;// canal G
		imagen[miPixel + 2] = 0; // canal B
		imagen[miPixel + 3] = 0; // canal alfa
	}
	else
	{
		imagen[miPixel + 0] = 255; // canal R
		imagen[miPixel + 1] = 255;// canal G
		imagen[miPixel + 2] = 255; // canal B
		imagen[miPixel + 3] = 0; // canal alfa
	}
}
// MAIN: rutina principal ejecutada en el host
int main(int argc, char** argv)
{

	// Busqueda de dispositivos
	int deviceCount;
	hipGetDeviceCount(&deviceCount);

	// Muestra información de los dispositivos encontrados
	if (deviceCount == 0)
	{
		printf("!!!!!No se han encontrado dispositivos CUDA!!!!!\n");
		printf("<pulsa [INTRO] para finalizar>");
		getchar();
		return 1;
	}
	else
	{
		// Muestra los datos de cada dispositivo encontrado
		for (int id = 0; id < deviceCount; id++)
		{
			propiedades_Device(id);
		}
	}

	// Declaracion del bitmap:
	// Inicializacion de la estructura RenderGPU
	RenderGPU foto(ANCHO, ALTO);

	// Tamaño del bitmap en bytes
	size_t bmp_size = foto.image_size();

	// Asignacion y reserva de la memoria en el host (framebuffer)
	unsigned char* host_bitmap = foto.get_ptr();

	// Reserva en el device
	unsigned char* dev_bitmap;
	hipMalloc((void**)&dev_bitmap, bmp_size);

	// Lanzamos un kernel bidimensional con bloques de 256 hilos (16x16)
	dim3 hilosB(NUMHILOS, NUMHILOS);

	// Calculamos el numero de bloques necesario (un hilo por cada pixel)
	dim3 Nbloques(ANCHO / NUMHILOS, ALTO / NUMHILOS);

	// Declaración del evento que calcula el tiempo de ejecución
	hipEvent_t start;
	hipEvent_t stop;

	// Creacion del evento
	hipEventCreate(&start);
	hipEventCreate(&stop);

	// Captura de la marca de tiempo de inicio
	hipEventRecord(start, 0);

	// Generamos el bitmap
	kernel << <Nbloques, hilosB >> > (dev_bitmap);

	// Captura el final de la marca de tiempo
	hipEventRecord(stop, 0);

	// Sincronizacion GPU-CPU
	hipEventSynchronize(stop);

	// Calculo del tiempo en milisegundos
	float elapsedTime;
	hipEventElapsedTime(&elapsedTime, start, stop);

	// Impresion de resultados
	printf("> Tiempo de ejecucion\t\t: %f ms\n", elapsedTime);
	printf("***************************************************\n");

	// Finalización del evento
	hipEventDestroy(start);
	hipEventDestroy(stop);

	// Copiamos los datos desde la GPU hasta el framebuffer para visualizarlos
	hipMemcpy(host_bitmap, dev_bitmap, bmp_size, hipMemcpyDeviceToHost);

	// Visualizacion y salida
	// La funcion ″display_and_exit()″ no retorna e impide continuar con el main()
	foto.display_and_exit();

	return 0;
}
